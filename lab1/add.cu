
#include <hip/hip_runtime.h>
#include <iostream>
#include <algorithm>

// SPMD -- single program multiple data
__global__ void add(float* v1, float* v2, int N) {
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x; // one thread 
    if (thread_id < N) {
        v1[thread_id] = v1[thread_id] + v2[thread_id];
    }
}

int main(int argc, char* argv[]) {
    const int N = (1 << 20);
    // std::array<float, N> vec1(1.0f), vec2(2.0f);
    float* h_vec1 = new float[N];
    float* h_vec2 = new float[N];
    float* d_vec1;
    float* d_vec2;
    hipMalloc(&d_vec1, N * sizeof(float));
    hipMalloc(&d_vec2, N * sizeof(float));
    for (int i = 0; i < N; ++i) {
        h_vec1[i] = 1.0f;
        h_vec2[i] = 2.0f;
    }

    hipMemcpy(d_vec1, h_vec1, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_vec2, h_vec2, N * sizeof(float), hipMemcpyHostToDevice);

    // cudaMallocMenaged(&h_vec1, N * sizeof(float));
    // cudaMallocMenaged(&h_vec1, N * sizeof(float));

    dim3 num_threads(1024);
    dim3 num_blocks(N / 1024 + 1);

    add<<<num_blocks, num_threads>>>(d_vec1, d_vec2, N);
    hipDeviceSynchronize();
    hipMemcpy(h_vec1, d_vec1, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i) {
        if (h_vec1[i] != 3) {
            std::cout << "Incorrect result!" << std::endl;
            return 1;
        }
    }
    delete[] h_vec1;
    delete[] h_vec2;
    hipFree(d_vec1);
    hipFree(d_vec2);
}

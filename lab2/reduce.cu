
#include <hip/hip_runtime.h>
#include <iostream>

/** 
@see https://developer.download.nvidia.com/assets/cuda/files/reduction.pdf
*/
__global__ void reduce0(int *in, int *out) {
    extern __shared__ int shm[]; // pamiec dzielona
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    shm[tid] = in[gid];
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        if (tid % (2 * stride) == 0) {
            shm[tid] += shm[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = shm[0];
    }
} // 208 mic sek

__global__ void reduce1(int *in, int *out) {
    extern __shared__ int shm[]; // pamiec dzielona
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    shm[tid] = in[gid];
    __syncthreads();

    for (unsigned int stride = 1; stride < blockDim.x; stride *= 2) {
        unsigned int index = 2 * stride * tid;
        if (index < blockDim.x) {
            shm[index] += shm[index + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = shm[0];
    }
} // 208 mic sek

//shared memory jest zorganizowana w 32 banki, w zaleznosci od precyzji moga byc wieksze lub mniejsze. Standardowe maja 32 bity. Konflikty w odczytywaniu tych bankow to Bank Conflicts. Elementy tabeli sa kazdy w innym banku, ale co 32 jest w tym samym, Odczytywanie z tego samego banku to konflikt. 

__global__ void reduce2(int *in, int *out) {
    extern __shared__ int shm[]; // pamiec dzielona
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    shm[tid] = in[gid];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0 ; stride /= 2) {
        if (tid < stride) {
            shm[tid] += shm[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = shm[0];
    }
} // 208 mic sek

__global__ void reduce3(int *in, int *out) {
    extern __shared__ int shm[]; // pamiec dzielona
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    shm[tid] = in[gid] + in[gid + blockDim.x];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 0 ; stride /= 2) {
        if (tid < stride) {
            shm[tid] += shm[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = shm[0];
    }
} // 210.94us

// 
// bez volatile jest optymalizacja i nie dziala
__device__ void warpReduce(volatile int *data, int tid) {
    data[tid] += data[tid + 32];
    data[tid] += data[tid + 16];
    data[tid] += data[tid + 8];
    data[tid] += data[tid + 4];
    data[tid] += data[tid + 2];
    data[tid] += data[tid + 1];
}

__global__ void reduce4(int *in, int *out) {
    extern __shared__ int shm[]; // pamiec dzielona
    int tid = threadIdx.x;
    int gid = blockIdx.x * blockDim.x + threadIdx.x;
    shm[tid] = in[gid] + in[gid + blockDim.x];
    __syncthreads();

    for (unsigned int stride = blockDim.x / 2; stride > 32 ; stride /= 2) {
        if (tid < stride) {
            shm[tid] += shm[tid + stride];
        }
        __syncthreads();
    }

    if (tid < 32) {
        warpReduce(shm, tid);
    }

    if (tid == 0) {
        out[blockIdx.x] = shm[0];
    }
} // 72.192us

template <unsigned int blockSize>
__device__ void warpReduce(volatile int* sdata, int tid) {
    if (blockSize >= 512) {
        if (tid < 256) { 
            sdata[tid] += sdata[tid + 256]; 
        } __syncthreads(); 
    }
    if (blockSize >= 256) {
        if (tid < 128) { 
            sdata[tid] += sdata[tid + 128]; 
        } __syncthreads(); 
    }
    if (blockSize >= 128) {
        if (tid <  64)  { 
            sdata[tid] += sdata[tid +   64]; 
        } __syncthreads(); 
    }
    if (tid < 32) warpReduce<blockSize>(sdata, tid);
    if (blockSize >= 64) sdata[tid] += sdata[tid + 32]; 
    if (blockSize >= 32) sdata[tid] += sdata[tid + 16]; 
    if (blockSize >= 16) sdata[tid] += sdata[tid +  8]; 
    if (blockSize >=   8) sdata[tid] += sdata[tid +  4]; 
    if (blockSize >=   4) sdata[tid] += sdata[tid +  2]; 
    if (blockSize >=   2) sdata[tid] += sdata[tid +  1];
}

int main(int argc, char **argv) {
    const int N = 1 << 20;
    int *h_data = new int[N];
    int *d_in, *d_out;

    hipMalloc(&d_in, N * sizeof(int));
    hipMalloc(&d_out, N * sizeof(int));

    hipMemset(d_out, 0, N * sizeof(int));

    for (int i = 0; i < N; ++i) {
        h_data[i] = 1;
    }

    hipMemcpy(d_in, h_data, N * sizeof(int), hipMemcpyHostToDevice);

    // reduce ....
    // int reduce_num = 5;
    int num_threads = 1024;

    // switch (reduce_num) {
    //     case 0: {
    //         int num_blocks = N / 1024;
    //         int shm_size = num_threads * sizeof(int);
    //         reduce0<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    //         reduce0<<<1, num_threads, shm_size>>>(d_out, d_out);
    //     }
    //     case 1: {
    //         int num_blocks = N / 1024;
    //         int shm_size = num_threads * sizeof(int);
    //         reduce1<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    //         reduce1<<<1, num_threads, shm_size>>>(d_out, d_out);
    //     }
    //     case 2: {
    //         int num_blocks = N / 1024;
    //         int shm_size = num_threads * sizeof(int);
    //         reduce2<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    //         reduce2<<<1, num_threads, shm_size>>>(d_out, d_out);
    //     }
    //     case 3: {
    //         int num_blocks = N / 1024 / 2;
    //         int shm_size = num_threads * sizeof(int);
    //         reduce3<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    //         reduce3<<<1, num_threads, shm_size>>>(d_out, d_out);
    //     }
    //     case 4: {
    //         int num_blocks = N / 1024 / 2;
    //         int shm_size = num_threads * sizeof(int);
    //         reduce4<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    //         reduce4<<<1, num_threads, shm_size>>>(d_out, d_out);
    //     }
    // }
    int num_blocks = N / 1024 / 2;
    int shm_size = num_threads * sizeof(int);
    reduce4<<<num_blocks, num_threads, shm_size>>>(d_in, d_out);
    reduce4<<<1, num_threads, shm_size>>>(d_out, d_out);

    hipMemcpy(h_data, d_out, N * sizeof(int), hipMemcpyDeviceToHost);

    if (h_data[0] != N) {
        std::cout << "incorrect result" << std::endl;
        return 1;
    }

    delete[] h_data;
    hipFree(d_in);
    hipFree(d_out);
    return 0;
}